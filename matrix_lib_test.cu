#include "hip/hip_runtime.h"
// Bernardo Bach - 1613231
// Eduardo Luna - 2111484


#include "matrix_lib.h"
#include "timer.h"
#include <stdio.h>
#include <stdlib.h>


typedef struct matrix Matrix;

float *matrix_from(char *from, int size) {
  float *matrix = aligned_alloc(32, sizeof(float) * size);
  FILE *f = fopen(from, "rb");
  fread(matrix, sizeof(float), size, f);

  fclose(f);
  return matrix;
}

void save_matrix(char *where, float *matrix, int size) {
  FILE *f = fopen(where, "w+b");
  int count;
  fwrite(matrix, sizeof(float), size, f);
  fclose(f);
  return;
}

void print_matrix(float *matrix_row, int row_size) {

  for (int count = 0; count < row_size; count++) {
    printf("%.2f ", matrix_row[count]);
    if ((count + 1) % 8 == 0)
      printf("\n");
    if (count == 255)
      break;
  }
  if (row_size > 256) {
    printf("=====  Print lenght limit of 256 values reached  =====\n");
  }
  return;
}

int check_matrix_result(Matrix *correct_m, Matrix *questionable_m, int row_len){
for(int count = 0; count < row_len; count++) {
  if(correct_m->rows[count] != questionable_m->rows[count]) {
    printf("Matrix is wrong\n\n");
    return 0;
  }
}
  printf("\nMatrix is correct!\n\n");
  return 1;
}

int check_input(int height_A, int width_A, int height_B, int width_B, int in_num_threads) {

  if (height_A % 8 != 0 || width_A % 8 != 0 || height_B % 8 != 0 || width_B % 8 != 0) {
    printf("Error: Invalid matrix size\n");
    return 0;
  }

  else if (width_A != height_B) {
    printf("Error: Matrices not compatible\n");
    return 0;
  }

  else if (height_A % in_num_threads != 0){
    printf("Error: Number of threads greater than matrix A height\n");
    return 0;
  }

  else return 1;

}

int main(int argc, char *argv[]) {

  if (!check_input(atoi(argv[2]), atoi(argv[3]), atoi(argv[4]), atoi(argv[5]), atoi(argv[6]))) return 0;
  
  int a_row_len, b_row_len, c_row_len, error_count = 0;
  float scalar_value;
  Matrix *matrixA, *matrixB, *matrixC, *matrix_check;
  struct timeval start, stop, overall_t1, overall_t2;

  gettimeofday(&overall_t1, NULL);


//adicionar inidices 7,8 e modificar  6

  set_number_threads(atoi(argv[6]));
  matrixA = malloc(sizeof(Matrix));
  matrixB = malloc(sizeof(Matrix));
  matrixC = malloc(sizeof(Matrix));
  matrix_check = malloc(sizeof(Matrix));

  scalar_value = atof(argv[1]);

  matrixA->height = atoi(argv[2]);
  matrixA->width = atoi(argv[3]);

  matrixB->height = atoi(argv[4]);
  matrixB->width = atoi(argv[5]);

  matrixC->height = matrixA->height;
  matrixC->width = matrixB->width;

  matrix_check->height = matrixA->height;
  matrix_check->width = matrixB->width;
  
  a_row_len = matrixA->height * matrixA->width;
  b_row_len = matrixB->height * matrixB->width;
  c_row_len = matrixC->height * matrixC->width;

  matrixA->h_rows = matrix_from(argv[9], a_row_len);
  
  if(hipMalloc(&matrixA->d_rows,sizeof(float) * a_row_len) != hipSuccess)
    printf("Cuda Malloc on array A  returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
  
  matrixB->h_rows = matrix_from(argv[10], b_row_len);
  
  if(hipMalloc(&matrixB->d_rows,sizeof(float) * b_row_len) != hipSuccess)
    printf("Cuda Malloc on array A  returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
  
  matrixC->h_rows = (float *)malloc(sizeof(float) * c_row_len);
  
  if(hipMalloc(&matrixC->d_rows,sizeof(float) * c_row_len) != hipSuccess)
    printf("Cuda Malloc on array A  returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
  
  matrix_check->h_rows = (float *)malloc(sizeof(float) * c_row_len);
  
  // printing all matrices
  printf("====== Matrix A ======\n");
  print_matrix(matrixA->rows_h, a_row_len);

  printf("\n====== Matrix B ======\n");
  print_matrix(matrixB->rows_h, b_row_len);

  printf("\n====== Matrix C ======\n");
  print_matrix(matrixC->rows_h, c_row_len);

  // executing and timing scalar_matrix_mult
  printf("\nExecuting scalar_matrix_mult . . . \n");

  gettimeofday(&start, NULL);
  error_count += scalar_matrix_mult(scalar_value, matrixA);
  gettimeofday(&stop, NULL);

  // printing scalar_matrix_mult result and time
  printf("====== Scalar * Matrix A ======\n");
  print_matrix(matrixA->h_rows, a_row_len);
  printf("\n===========================================\nscalar_matrix_mult elapsed time: %.4f ms\n===========================================\n\n\n",
  timedifference_msec(start, stop));

  // executing and timing matrix_matrix_mult
  printf("Executing matrix_matrix_mult . . .\n");

  gettimeofday(&start, NULL);
  error_count += matrix_matrix_mult(matrixA, matrixB, matrixC);
  gettimeofday(&stop, NULL);

  // printing matrix_matrix_mult and time
  printf("====== MatrixA * MatrixB  ======\n");
  print_matrix(matrixC->h_rows, c_row_len);
  printf("\n===============================================\nmatrix_matrix_mult elapsed time: %.4f ms\n===============================================\n\n\n",
  timedifference_msec(start, stop));

  save_matrix(argv[9], matrixA->h_rows, a_row_len);
  save_matrix(argv[10], matrixC->h_rows, c_row_len);
  
  printf("Checking matrix . . .\n");
  //avx_matrix_matrix_mult(matrixA, matrixB, matrix_check);
  matrix_matrix_mult(matrixA, matrixB, matrix_check);
  error_count += check_matrix_result(matrixC, matrix_check, c_row_len);
  
  error_count = abs(error_count - 3);
  printf("====================\n Errors detected: %d\n====================\n", error_count);

  free(matrixA->h_rows);
  free(matrixB->h_rows);
  free(matrixC->h_rows);
  free(matrix_check->h_rows);
  
  freeCuda(matrixA->d_rows);
  freeCuda(matrixB->d_rows);
  freeCuda(matrixC->d_rows);
  
  free(matrixA);
  free(matrixB);
  free(matrixC);
  free(matrix_check);

  gettimeofday(&overall_t2, NULL);
  printf("Overall time: %.4f ms\n", timedifference_msec(overall_t1, overall_t2));

  return 0;
}